#include "hip/hip_runtime.h"
/**
	C++ client example using sockets
*/
#include<iostream>	//cout
#include<stdio.h>	//printf
#include<string.h>	//strlen
#include<string>	//string
#include<cstring>
#include<sys/socket.h>	//socket
#include<arpa/inet.h>	//inet_addr
#include<netdb.h>	//hostent
#include<stdlib.h>
#include "allConstant.h"
#include <unistd.h>
#include <thread>
#include <chrono> 
#include "safeQueue.h"
#include "quartic.h"
#include <map>
#include <sstream> 
#include <iterator>
#include <unordered_set>
#include <vector>
#include <math.h>
#include <future>
#include <complex>
#include <algorithm>
#include "Point.h"
#include "CycleTrial.h"
#include "HyperTrial.h"
#include "generalFunction.h"
#include "userClass.h"
#include "EllipseTrial.h"


using namespace std;
map<string,int> indexMap;
User allUser[userSize];
string idList[userSize];
int userNow=-1;
int getUser(string tmpID);
void setupUser(string tmpID,int epoch,int nodeNum,int anchorNum,int radioRange);
void configNodes(string tmpID,int epoch,int nodeID,int X,int Y);
void addTrajectory(string tmpID,int epoch,int traInd,int X,int Y);

__global__ void goOver3(int n, ellipseTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))+sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }

        float rate3 = data[k].rate3;
        data[k].grAr=rate3*total;
        data[k].acAr = total;

    }
}

__global__ void bestEllipse(int n, ellipseTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))+sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }
        float rate = total / data[k].tArea;
        float rate3 = rate*rate*rate;
        if ( rate3<1.1){
            data[k].grAr=rate3*total;
            data[k].acAr = total;
            data[k].rate3 = rate3;
        }
    }
}




__global__ void goOver2(int n, hyperTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))-sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }

        float rate3 = data[k].rate3;
        data[k].grAr=rate3*total;
        data[k].acAr = total;

    }
}

__global__ void bestHyper(int n, hyperTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))-sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }
        float rate = total / (data[k].tArea +0.1);
        float rate3 = rate*rate*rate;
        if ( rate3<1.1){
            data[k].grAr=rate3*total;
            data[k].acAr = total;
            data[k].rate3 = rate3;
        }
    }
}

__global__ void bestTwoCycle(int n, twoCycleTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x1=data[k].c1X;
        float y1=data[k].c1Y;
        float x2=data[k].c2X;
        float y2=data[k].c2Y;
        float r1=data[k].h1*data[k].d;
        float r2=data[k].h2*data[k].d;
        float r3=r1-data[k].d;
        float rr3=r3*r3;
        float rr1=r1*r1;
        float rr2=r2*r2;
        float total=0.0;
        for(int l=0;l<m;){
            float i=area[l++];
            float j=area[l++];
            float di1=x1-i;
            float dj1=y1-j;
            float di2=x2-i;
            float dj2=y2-j;
            if (di1*di1+dj1*dj1<=rr1 && di1*di1+dj1*dj1>rr3 && di2*di2+dj2*dj2<=rr2)
                total+=1.0;
        }
        float rate=total / (data[k].tArea+0.1);
        float rate3=rate*rate*rate;
        if (rate3<1.1){
                
            data[k].grAr=rate3*total ; 
            data[k].acAr=total;
            data[k].rate3=rate3;
        }
    }    

}

__global__ void goOver1(int n, twoCycleTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x1=data[k].c1X;
        float y1=data[k].c1Y;
        float x2=data[k].c2X;
        float y2=data[k].c2Y;
        float r1=data[k].h1*data[k].d;
        float r2=data[k].h2*data[k].d;
        float r3=r1-data[k].d;
        float rr3=r3*r3;
        float rr1=r1*r1;
        float rr2=r2*r2;
        float total=0.0;
        for(int l=0;l<m;){
            float i=area[l++];
            float j=area[l++];
            float di1=x1-i;
            float dj1=y1-j;
            float di2=x2-i;
            float dj2=y2-j;
            if (di1*di1+dj1*dj1<=rr1 && di1*di1+dj1*dj1>rr3 && di2*di2+dj2*dj2<=rr2)
                total+=1.0;
        }

        float rate3=data[k].rate3;

        data[k].grAr=rate3*total ; 
        data[k].acAr=total;

    }    

}


int getUser(string tmpID){
    
    if(indexMap.find(tmpID)==indexMap.end()){
        
        User curUser=User(tmpID);
        userNow=(userNow+1)%userSize;
        cout<<"Setting up a new User :"<<tmpID<<" at: "<<userNow<<endl;
        if(idList[userNow].length()>0)indexMap.erase(idList[userNow]);
        idList[userNow]=tmpID;
        indexMap.insert(make_pair(tmpID,userNow));
        allUser[userNow]=curUser;
        return userNow;
    }
    else{
        return  indexMap[tmpID];
    
    }
}

void setupUser(string tmpID,int epoch,int nodeNum,int anchorNum,int radioRange)
{
    User& curUser = allUser[getUser(tmpID)];
    //cout<<epoch<<" : "<<curUser.getEpoch()<<endl;
    if(epoch >= curUser.getEpoch())
    {
        curUser.setNodes(nodeNum);
        curUser.setAnchor(anchorNum);
        curUser.setEpoch(epoch);
        curUser.setRange(radioRange);
        cout<<"setting up user: "<<  tmpID <<" in: "<<getUser(tmpID)<<endl;
    }
}

void addTrajectory(string tmpID,int epoch,int traInd, int X,int Y)
{
    User& curUser = allUser[getUser(tmpID)];
    //cout<<epoch<<" : "<<curUser.getEpoch()<<endl;
    if(epoch >= curUser.getEpoch())
    {
        curUser.setEpoch(epoch);
        curUser.setTraj(X,Y,traInd);
        
    }
}


void tcp_client::setupTAS(string tmpID,int epoch, string requestID, int totalStroke)
{
    //cout<<"setting TAS"<<endl;
    User& curUser = allUser[getUser(tmpID)];
    //cout<<epoch<<" : "<<curUser.getEpoch()<<endl;
    if(epoch >= curUser.getEpoch() && curUser.resultMap.find(requestID)==curUser.resultMap.end())
    {
      //cout<<"push to computing Queue for : " <<tmpID+" "+requestID<<endl;
      curUser.resultMap.insert(make_pair(requestID,""));
      curUser.genTAS(totalStroke);
      computingQueue.push(tmpID+" "+requestID);
      //cout<<"push to computing Queue for : " <<tmpID+" "+requestID<<endl;
      //curUser.printArea();
      
    }
    else
    {
      cout<<"The computation requestion already in processing"<<endl;
    
    }
}


void configNodes(string tmpID,int epoch,int nodeID,int X,int Y)
{
    User& curUser = allUser[getUser(tmpID)];
    if(epoch >= curUser.getEpoch())
    {
        curUser.setX(nodeID,X);
        curUser.setY(nodeID,Y);
    }  
}

string getRoutingMSG2(string userRequest)
{
    string res="";
    stringstream ur(userRequest);
    string tmpID;
    string requestID;
    ur>>tmpID;
    ur>>requestID;
    User& curUser = allUser[getUser(tmpID)];
    
    float *TAS,*d_TAS;
    int tasSize=curUser.TAS.size();
    TAS=(float*)malloc(sizeof(float)*tasSize*2);
    hipMalloc((void**)&d_TAS, sizeof(float) *tasSize*2);
    int counter=0;
    for(string t:curUser.TAS)
    {
        stringstream tt(t);
        float x,y;
        tt>>x;
        tt>>y;
        TAS[counter++]=x;
        TAS[counter++]=y;
    }
    hipMemcpy(d_TAS, TAS, sizeof(float) *tasSize*2, hipMemcpyHostToDevice);
    
    //cout<<"I'm OK Here"<<endl;
    short hv[nodeSize][anchorSize];
    curUser.getHopInfo(hv);
    vector<ellipseTrial> ellipseTrials=curUser.findEllipseTrial(hv);
    counter=0; 
    ellipseTrial *eTri;
    ellipseTrial *d_eTri;
    if(!ellipseTrials.empty())
    {
        eTri=(ellipseTrial*)malloc(sizeof(ellipseTrial)*ellipseTrials.size());
        
        //cout<<"I'm OK after here"<<endl;
        for(ellipseTrial et: ellipseTrials)
        {
            eTri[counter++]=et;
        }
        hipMalloc((void**)&d_eTri, sizeof(ellipseTrial) * ellipseTrials.size());
        hipMemcpy(d_eTri,eTri,sizeof(ellipseTrial) * ellipseTrials.size(),hipMemcpyHostToDevice);
        cout<<"finish copy totoal trial: "<<ellipseTrials.size()<<endl;        
        bestEllipse<<<2048,256>>>(ellipseTrials.size(),d_eTri,d_TAS,tasSize*2);
       
        //hipFree(d_cTri);
    }
    vector<hyperTrial> hyperTrials=curUser.findHyperTrial(hv);   
    //cout<<"number of hyperTrial: "<<hyperTrials.size()<<endl;
    hyperTrial *hTri;
    hyperTrial *d_hTri;
    int counter2=0;
    hipDeviceSynchronize();
    if(!hyperTrials.empty())
    {
        hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*hyperTrials.size());

        //cout<<"I'm OK after here"<<endl;
        for(hyperTrial ht: hyperTrials)
        {
            hTri[counter2++]=ht;
        }
        hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *hyperTrials.size());        
        hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyHostToDevice);
        cout<<"finish copy totoal trial: "<<hyperTrials.size()<<endl;
        cout<<"********TAS size is: "<<tasSize<<"******"<<endl;
        bestHyper<<<2048,256>>>(hyperTrials.size(),d_hTri,d_TAS,tasSize*2);
        //hipDeviceSynchronize();
        
        //hipFree(d_hTri);
    }
    
    if(ellipseTrials.empty() || hyperTrials.empty())return "No result";
    hipDeviceSynchronize();
    hipMemcpy(eTri,d_eTri,sizeof(ellipseTrial) *ellipseTrials.size(),hipMemcpyDeviceToHost); 
    hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyDeviceToHost);  
      
    cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<tasSize<<endl;
    res+=findBestTry2(eTri,hTri, counter, counter2, curUser.TAS);
    
    sort(eTri, eTri+counter, sortEllipseTrial);
    sort(hTri, hTri+counter2, sortHyperTrial);
    counter = 5000000 < counter ? 5000000 : counter;
    counter2 = 5000000 < counter2 ? 5000000 : counter2;
    
    
    //hipMalloc((void**)&d_cTri, sizeof(twoCycleTrial) *counter);
    hipMemcpy(d_eTri,eTri,sizeof(ellipseTrial) *counter, hipMemcpyHostToDevice);
    //hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *counter2);  
    hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *counter2, hipMemcpyHostToDevice);
    //cout<<"copy to device again"<<endl;
    int newSize = curUser.TAS.size();
    cout<<"new TAS SIZE is: "<<newSize<<"; old TAS size is"<<tasSize<<endl;
    while(newSize>0.15*tasSize)
    {
      
      newSize=curUser.TAS.size();
      cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<newSize<<endl;
      int tasInd=0;
      for(string t:curUser.TAS)
      {
          stringstream tt(t);
          float x,y;
          tt>>x;
          tt>>y;
          TAS[tasInd++]=x;
          TAS[tasInd++]=y;
      }
      //hipFree(d_TAS);
      //hipMalloc((void**)&d_TAS, sizeof(float) *newSize*2);
      hipMemcpy(d_TAS, TAS, sizeof(float) *newSize*2, hipMemcpyHostToDevice);
      goOver3<<<2048,256>>>(counter, d_eTri, d_TAS, newSize*2);
      goOver2<<<2048,256>>>(counter2, d_hTri, d_TAS, newSize*2);
      hipDeviceSynchronize();
      
      //free(cTri);
      //free(hTri);
      //cTri=(twoCycleTrial*)malloc(sizeof(twoCycleTrial)*counter);
      //hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*counter2);
      hipMemcpy(eTri,d_eTri,sizeof(ellipseTrial) *counter,hipMemcpyDeviceToHost);  
      hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *counter2,hipMemcpyDeviceToHost);
      res+=findBestTry2(eTri,hTri, counter, counter2, curUser.TAS);
    }
    
    hipFree(d_eTri);
    hipFree(d_hTri);
    hipFree(d_TAS);
    free(TAS);
    free(eTri); 
    free(hTri);
    
    cout<<"sending back result!!!!!!!!!!!!!!!"<<endl;     
    return res;

}


string getRoutingMSG(string userRequest)
{
    string res="";
    stringstream ur(userRequest);
    string tmpID;
    string requestID;
    ur>>tmpID;
    ur>>requestID;
    User& curUser = allUser[getUser(tmpID)];
    
    float *TAS,*d_TAS;
    int tasSize=curUser.TAS.size();
    TAS=(float*)malloc(sizeof(float)*tasSize*2);
    hipMalloc((void**)&d_TAS, sizeof(float) *tasSize*2);
    int counter=0;
    for(string t:curUser.TAS)
    {
        stringstream tt(t);
        float x,y;
        tt>>x;
        tt>>y;
        TAS[counter++]=x;
        TAS[counter++]=y;
    }
    hipMemcpy(d_TAS, TAS, sizeof(float) *tasSize*2, hipMemcpyHostToDevice);
    
    //cout<<"I'm OK Here"<<endl;
    short hv[nodeSize][anchorSize];
    curUser.getHopInfo(hv);
    vector<twoCycleTrial> cycleTrials=curUser.findTwoCycleTrial(hv);
    counter=0; 
    twoCycleTrial *cTri;
    twoCycleTrial *d_cTri;
    if(!cycleTrials.empty())
    {
        cTri=(twoCycleTrial*)malloc(sizeof(twoCycleTrial)*cycleTrials.size());
        
        //cout<<"I'm OK after here"<<endl;
        for(twoCycleTrial ct: cycleTrials)
        {
            cTri[counter++]=ct;
        }
        hipMalloc((void**)&d_cTri, sizeof(twoCycleTrial) *cycleTrials.size());
        hipMemcpy(d_cTri,cTri,sizeof(twoCycleTrial) *cycleTrials.size(),hipMemcpyHostToDevice);
        cout<<"finish copy totoal trial: "<<cycleTrials.size()<<endl;        
        bestTwoCycle<<<2048,256>>>(cycleTrials.size(),d_cTri,d_TAS,tasSize*2);
       
        //hipFree(d_cTri);
    }
    vector<hyperTrial> hyperTrials=curUser.findHyperTrial(hv);   
    //cout<<"number of hyperTrial: "<<hyperTrials.size()<<endl;
    hyperTrial *hTri;
    hyperTrial *d_hTri;
    int counter2=0;
    if(!hyperTrials.empty())
    {
        hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*hyperTrials.size());

        //cout<<"I'm OK after here"<<endl;
        for(hyperTrial ht: hyperTrials)
        {
            hTri[counter2++]=ht;
        }
        hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *hyperTrials.size());        
        hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyHostToDevice);
        cout<<"finish copy totoal trial: "<<hyperTrials.size()<<endl;
        cout<<"********TAS size is: "<<tasSize<<"******"<<endl;
        bestHyper<<<2048,256>>>(hyperTrials.size(),d_hTri,d_TAS,tasSize*2);
        //hipDeviceSynchronize();
        
        //hipFree(d_hTri);
    }
    
    if(cycleTrials.empty() || hyperTrials.empty())return "No result";
    hipDeviceSynchronize();
    hipMemcpy(cTri,d_cTri,sizeof(twoCycleTrial) *cycleTrials.size(),hipMemcpyDeviceToHost); 
    hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyDeviceToHost);  
      
    cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<tasSize<<endl;
    res+=findBestTry(cTri,hTri, counter, counter2, curUser.TAS);
    
    sort(cTri, cTri+counter, sortCycleTrial);
    sort(hTri, hTri+counter2, sortHyperTrial);
    counter = 1000000 < counter ? 1000000 : counter;
    counter2 = 5000000 < counter2 ? 5000000 : counter2;
    
    
    //hipMalloc((void**)&d_cTri, sizeof(twoCycleTrial) *counter);
    hipMemcpy(d_cTri,cTri,sizeof(twoCycleTrial) *counter, hipMemcpyHostToDevice);
    //hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *counter2);  
    hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *counter2, hipMemcpyHostToDevice);

    int newSize = 0;
    do
    {
      
      newSize=curUser.TAS.size();
      cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<newSize<<endl;
      int tasInd=0;
      for(string t:curUser.TAS)
      {
          stringstream tt(t);
          float x,y;
          tt>>x;
          tt>>y;
          TAS[tasInd++]=x;
          TAS[tasInd++]=y;
      }
      //hipFree(d_TAS);
      //hipMalloc((void**)&d_TAS, sizeof(float) *newSize*2);
      hipMemcpy(d_TAS, TAS, sizeof(float) *newSize*2, hipMemcpyHostToDevice);
      goOver1<<<2048,256>>>(counter, d_cTri, d_TAS, newSize*2);
      goOver2<<<2048,256>>>(counter2, d_hTri, d_TAS, newSize*2);
      hipDeviceSynchronize();
      
      //free(cTri);
      //free(hTri);
      //cTri=(twoCycleTrial*)malloc(sizeof(twoCycleTrial)*counter);
      //hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*counter2);
      hipMemcpy(cTri,d_cTri,sizeof(twoCycleTrial) *counter,hipMemcpyDeviceToHost);  
      hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *counter2,hipMemcpyDeviceToHost);
      res+=findBestTry(cTri,hTri, counter, counter2, curUser.TAS);
    }while(newSize>0.15*tasSize);
    
    hipFree(d_cTri);
    hipFree(d_hTri);
    hipFree(d_TAS);
    free(TAS);
    free(cTri); 
    free(hTri);     
    return res;

}


void tcp_client::addData(string data)
{
    int begin=data.find("*,");
    int end=data.find(",*",begin+1);
    //cout<<data<<endl;
    if(begin>=0&&end>begin)
    {
        string realData=data.substr(begin+2,end-begin-2);

        int length=realData.length();
        int comma=0;
        for(int i=0;i<length;i++){
            if(realData[i]==',')comma++;
        }
        //cout<<realData<<endl;
        if(comma==5){
            int first=realData.find(",");
            string tmpID=realData.substr(0,first);
            //cout<<tmpID.length()<<endl;
            if(tmpID.length()!=8 || tmpID.find("0.") == string::npos )return;
            
            int second=realData.find(",",first+1);
            int third=realData.find(",",second+1);
            int fourth=realData.find(",",third+1);
            int fifth=realData.find(",",fourth+1);
            
            int epoch=myStoi(realData.substr(first+1,second-first));
            int mod=myStoi(realData.substr(second+1,third-second));
            //cout<<mod<<endl;
            if(mod==0){
            
                int nodeNum=myStoi(realData.substr(third+1,fourth-third));
                int anchorNum=myStoi(realData.substr(fourth+1,fifth-fourth));
                int radioRange=myStoi(realData.substr(fifth+1,length-fourth));
                //cout<<"0: "<<tmpID<<":"<<epoch<<endl;
              if(nodeNum<nodeSize&&anchorNum<anchorSize&&radioRange<=200&&nodeNum>0&&anchorNum>0&&radioRange>0)
                setupUser(tmpID,epoch,nodeNum,anchorNum,radioRange);
            }
            if(mod==1){
                int nodeID=myStoi(realData.substr(third+1,fourth-third));
                int X=myStoi(realData.substr(fourth+1,fifth-fourth));
                int Y=myStoi(realData.substr(fifth+1,length-fourth));
              if(nodeID<nodeSize&&nodeID>=0)  
                configNodes(tmpID,epoch,nodeID,X,Y);
            }
            if(mod==3){
                int traInd=myStoi(realData.substr(third+1,fourth-third));
                int X=myStoi(realData.substr(fourth+1,fifth-fourth));
                int Y=myStoi(realData.substr(fifth+1,length-fourth));
              if(traInd<strokeSize&&traInd>=0)
                addTrajectory(tmpID,epoch,traInd,X,Y);
            }
            if(mod==4){
                //cout<<"third: "<<third<<"fourth: "<<fourth<<endl;
                string requestID=realData.substr(third+1,fourth-third-1);
                int totalStroke=myStoi(realData.substr(fourth+1,fifth-fourth));
                if(totalStroke<strokeSize)
                {
                    cout<<"request realDATA "<<realData<<endl;
                    cout<<"here is the requestID "<<requestID<<endl;
                    setupTAS(tmpID,epoch,requestID, totalStroke);
                }
            
            }
        }

    }


}


int main(int argc , char *argv[])
{  
  srand(time(NULL));


	return 0;
}
 
 
 
