#include "hip/hip_runtime.h"
#include<iostream>	//cout
#include<stdio.h>	//printf
#include<string.h>	//strlen
#include<string>	//string
#include<cstring>
#include<sys/socket.h>	//socket
#include<arpa/inet.h>	//inet_addr
#include<netdb.h>	//hostent
#include<stdlib.h>
#include "allConstant.h"
#include <unistd.h>
#include <thread>
#include <chrono> 
#include "safeQueue.h"
#include "quartic.h"
#include <map>
#include <sstream> 
#include <iterator>
#include <unordered_set>
#include <vector>
#include <math.h>
#include <future>
#include <complex>
#include <algorithm>
#include "Point.h"
#include "CycleTrial.h"
#include "HyperTrial.h"
#include "generalFunction.h"
#include "userClass.h"
#include "EllipseTrial.h"

__global__ void goOver3(int n, ellipseTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))+sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }

        float rate3 = data[k].rate3;
        data[k].grAr=rate3*total;
        data[k].acAr = total;

    }
}

__global__ void bestEllipse(int n, ellipseTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))+sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }
        float rate = total / data[k].tArea;
        float rate3 = rate*rate*rate;
        if ( rate3<1.1){
            data[k].grAr=rate3*total;
            data[k].acAr = total;
            data[k].rate3 = rate3;
        }
    }
}




__global__ void goOver2(int n, hyperTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))-sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }

        float rate3 = data[k].rate3;
        data[k].grAr=rate3*total;
        data[k].acAr = total;

    }
}

__global__ void bestHyper(int n, hyperTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x = data[k].c3X;
        float y = data[k].c3Y;
        float r = data[k].h3 * data[k].avgD2;
        float a = data[k].ah * data[k].avgD1;
        
        float h1x = data[k].c1X;
        float h1y = data[k].c1Y;
        float h2x = data[k].c2X;
        float h2y = data[k].c2Y;
        
        float rr = r*r;
        float a2 = (data[k].ah-1.0)*data[k].avgD1;
        float total =0.0;
        
        for(int l = 0; l<m;){
            float i = area[l++];
            float j = area[l++];
            float tt = sqrtf((i-h1x)*(i-h1x)+(j-h1y)*(j-h1y))-sqrtf((i-h2x)*(i-h2x)+(j-h2y)*(j-h2y));
            float di = x-i;
            float dj = y-j;
            if(di * di + dj * dj <= rr && tt <= 2*a && tt >= 2* a2) total+=1.0;
        }
        float rate = total / (data[k].tArea +0.1);
        float rate3 = rate*rate*rate;
        if ( rate3<1.1){
            data[k].grAr=rate3*total;
            data[k].acAr = total;
            data[k].rate3 = rate3;
        }
    }
}

__global__ void bestTwoCycle(int n, twoCycleTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x1=data[k].c1X;
        float y1=data[k].c1Y;
        float x2=data[k].c2X;
        float y2=data[k].c2Y;
        float r1=data[k].h1*data[k].d;
        float r2=data[k].h2*data[k].d;
        float r3=r1-data[k].d;
        float rr3=r3*r3;
        float rr1=r1*r1;
        float rr2=r2*r2;
        float total=0.0;
        for(int l=0;l<m;){
            float i=area[l++];
            float j=area[l++];
            float di1=x1-i;
            float dj1=y1-j;
            float di2=x2-i;
            float dj2=y2-j;
            if (di1*di1+dj1*dj1<=rr1 && di1*di1+dj1*dj1>rr3 && di2*di2+dj2*dj2<=rr2)
                total+=1.0;
        }
        float rate=total / (data[k].tArea+0.1);
        float rate3=rate*rate*rate;
        if (rate3<1.1){
                
            data[k].grAr=rate3*total ; 
            data[k].acAr=total;
            data[k].rate3=rate3;
        }
    }    

}

__global__ void goOver1(int n, twoCycleTrial *data, float *area,int m){
    int index = threadIdx.x+blockIdx.x*blockDim.x;
    int stride=blockDim.x*gridDim.x;
    for(int k=index;k<n;k+=stride){
        float x1=data[k].c1X;
        float y1=data[k].c1Y;
        float x2=data[k].c2X;
        float y2=data[k].c2Y;
        float r1=data[k].h1*data[k].d;
        float r2=data[k].h2*data[k].d;
        float r3=r1-data[k].d;
        float rr3=r3*r3;
        float rr1=r1*r1;
        float rr2=r2*r2;
        float total=0.0;
        for(int l=0;l<m;){
            float i=area[l++];
            float j=area[l++];
            float di1=x1-i;
            float dj1=y1-j;
            float di2=x2-i;
            float dj2=y2-j;
            if (di1*di1+dj1*dj1<=rr1 && di1*di1+dj1*dj1>rr3 && di2*di2+dj2*dj2<=rr2)
                total+=1.0;
        }

        float rate3=data[k].rate3;

        data[k].grAr=rate3*total ; 
        data[k].acAr=total;

    }    

}

string getRoutingMSG_without_ellipse(User & curUser,vector<twoCycleTrial> & cycleTrials,vector<hyperTrial> & hyperTrials)
{
  string res = "";
  float *TAS,*d_TAS;
  int tasSize=curUser.TAS.size();
  TAS=(float*)malloc(sizeof(float)*tasSize*2);
  hipMalloc((void**)&d_TAS, sizeof(float) *tasSize*2);
  int counter=0;
  for(string t:curUser.TAS)
  {
      stringstream tt(t);
      float x,y;
      tt>>x;
      tt>>y;
      TAS[counter++]=x;
      TAS[counter++]=y;
  }
  hipMemcpy(d_TAS, TAS, sizeof(float) *tasSize*2, hipMemcpyHostToDevice);
  
  //cout<<"I'm OK Here"<<endl;
  counter=0; 
  twoCycleTrial *cTri;
  twoCycleTrial *d_cTri;
  if(!cycleTrials.empty())
  {
      cTri=(twoCycleTrial*)malloc(sizeof(twoCycleTrial)*cycleTrials.size());
      
      //cout<<"I'm OK after here"<<endl;
      for(twoCycleTrial ct: cycleTrials)
      {
          cTri[counter++]=ct;
      }
      hipMalloc((void**)&d_cTri, sizeof(twoCycleTrial) *cycleTrials.size());
      hipMemcpy(d_cTri,cTri,sizeof(twoCycleTrial) *cycleTrials.size(),hipMemcpyHostToDevice);
      cout<<"finish copy totoal trial: "<<cycleTrials.size()<<endl;        
      bestTwoCycle<<<2048,256>>>(cycleTrials.size(),d_cTri,d_TAS,tasSize*2);
     
      //hipFree(d_cTri);
  } 
  //cout<<"number of hyperTrial: "<<hyperTrials.size()<<endl;
  hyperTrial *hTri;
  hyperTrial *d_hTri;
  int counter2=0;
  if(!hyperTrials.empty())
  {
      hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*hyperTrials.size());

      //cout<<"I'm OK after here"<<endl;
      for(hyperTrial ht: hyperTrials)
      {
          hTri[counter2++]=ht;
      }
      hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *hyperTrials.size());        
      hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyHostToDevice);
      cout<<"finish copy totoal trial: "<<hyperTrials.size()<<endl;
      cout<<"********TAS size is: "<<tasSize<<"******"<<endl;
      bestHyper<<<2048,256>>>(hyperTrials.size(),d_hTri,d_TAS,tasSize*2);
      //hipDeviceSynchronize();
      
      //hipFree(d_hTri);
  }
  
  if(cycleTrials.empty() || hyperTrials.empty())return "No result";
  hipDeviceSynchronize();
  hipMemcpy(cTri,d_cTri,sizeof(twoCycleTrial) *cycleTrials.size(),hipMemcpyDeviceToHost); 
  hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyDeviceToHost);  
    
  cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<tasSize<<endl;
  res+=findBestTry(cTri,hTri, counter, counter2, curUser.TAS);
  
  sort(cTri, cTri+counter, sortCycleTrial);
  sort(hTri, hTri+counter2, sortHyperTrial);
  counter = 1000000 < counter ? 1000000 : counter;
  counter2 = 5000000 < counter2 ? 5000000 : counter2;
  
  
  //hipMalloc((void**)&d_cTri, sizeof(twoCycleTrial) *counter);
  hipMemcpy(d_cTri,cTri,sizeof(twoCycleTrial) *counter, hipMemcpyHostToDevice);
  //hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *counter2);  
  hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *counter2, hipMemcpyHostToDevice);

  int newSize = 0;
  int ccc = 0;
  do
  {
    ccc++;
    newSize=curUser.TAS.size();
    cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<newSize<<endl;
    int tasInd=0;
    for(string t:curUser.TAS)
    {
        stringstream tt(t);
        float x,y;
        tt>>x;
        tt>>y;
        TAS[tasInd++]=x;
        TAS[tasInd++]=y;
    }
    //hipFree(d_TAS);
    //hipMalloc((void**)&d_TAS, sizeof(float) *newSize*2);
    hipMemcpy(d_TAS, TAS, sizeof(float) *newSize*2, hipMemcpyHostToDevice);
    goOver1<<<2048,256>>>(counter, d_cTri, d_TAS, newSize*2);
    goOver2<<<2048,256>>>(counter2, d_hTri, d_TAS, newSize*2);
    hipDeviceSynchronize();
    
    //free(cTri);
    //free(hTri);
    //cTri=(twoCycleTrial*)malloc(sizeof(twoCycleTrial)*counter);
    //hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*counter2);
    hipMemcpy(cTri,d_cTri,sizeof(twoCycleTrial) *counter,hipMemcpyDeviceToHost);  
    hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *counter2,hipMemcpyDeviceToHost);
    res+=findBestTry(cTri,hTri, counter, counter2, curUser.TAS);
  }while(newSize>0.1*tasSize && ccc<30);
  
  hipFree(d_cTri);
  hipFree(d_hTri);
  hipFree(d_TAS);
  free(TAS);
  free(cTri); 
  free(hTri);     
  return res;
  
}

string getRoutingMSG_with_ellipse(User & curUser,vector<ellipseTrial> ellipseTrials,vector<hyperTrial> & hyperTrials)
{
  string res="";
  float *TAS,*d_TAS;
  int tasSize=curUser.TAS.size();
  TAS=(float*)malloc(sizeof(float)*tasSize*2);
  hipMalloc((void**)&d_TAS, sizeof(float) *tasSize*2);
  int counter=0;
  for(string t:curUser.TAS)
  {
      stringstream tt(t);
      float x,y;
      tt>>x;
      tt>>y;
      TAS[counter++]=x;
      TAS[counter++]=y;
  }
  hipMemcpy(d_TAS, TAS, sizeof(float) *tasSize*2, hipMemcpyHostToDevice);
  
  counter=0; 
  ellipseTrial *eTri;
  ellipseTrial *d_eTri;
  if(!ellipseTrials.empty())
  {
      eTri=(ellipseTrial*)malloc(sizeof(ellipseTrial)*ellipseTrials.size());
      
      //cout<<"I'm OK after here"<<endl;
      for(ellipseTrial et: ellipseTrials)
      {
          eTri[counter++]=et;
      }
      hipMalloc((void**)&d_eTri, sizeof(ellipseTrial) * ellipseTrials.size());
      hipMemcpy(d_eTri,eTri,sizeof(ellipseTrial) * ellipseTrials.size(),hipMemcpyHostToDevice);
      cout<<"finish copy totoal trial: "<<ellipseTrials.size()<<endl;        
      bestEllipse<<<2048,256>>>(ellipseTrials.size(),d_eTri,d_TAS,tasSize*2);
     
      
      
  }
  hipDeviceSynchronize();
  hipMemcpy(eTri,d_eTri,sizeof(ellipseTrial) *ellipseTrials.size(),hipMemcpyDeviceToHost); 
  hipDeviceSynchronize();
  hipFree(d_eTri);
  
  
  hyperTrial *hTri;
  hyperTrial *d_hTri;
  int counter2=0;
  
  if(!hyperTrials.empty())
  {
      hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*hyperTrials.size());

      //cout<<"I'm OK after here"<<endl;
      for(hyperTrial ht: hyperTrials)
      {
          hTri[counter2++]=ht;
      }
      hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *hyperTrials.size());        
      hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyHostToDevice);
      cout<<"finish copy totoal trial: "<<hyperTrials.size()<<endl;
      cout<<"********TAS size is: "<<tasSize<<"******"<<endl;
      bestHyper<<<2048,256>>>(hyperTrials.size(),d_hTri,d_TAS,tasSize*2);
      //hipDeviceSynchronize();
      
      //hipFree(d_hTri);
  }
  
  if(ellipseTrials.empty() || hyperTrials.empty())return "No result";
  hipDeviceSynchronize();
  hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *hyperTrials.size(),hipMemcpyDeviceToHost);  
  hipDeviceSynchronize();
  hipFree(d_hTri);
  
    
  cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<tasSize<<endl;
  res+=findBestTry2(eTri,hTri, counter, counter2, curUser.TAS);
  
  sort(eTri, eTri+counter, sortEllipseTrial);
  sort(hTri, hTri+counter2, sortHyperTrial);
  counter = 5000000 < counter ? 5000000 : counter;
  counter2 = 5000000 < counter2 ? 5000000 : counter2;
  
  
  hipMalloc((void**)&d_eTri, sizeof(ellipseTrial) *counter);
  hipMemcpy(d_eTri,eTri,sizeof(ellipseTrial) *counter, hipMemcpyHostToDevice);
  hipMalloc((void**)&d_hTri, sizeof(hyperTrial) *counter2);  
  hipMemcpy(d_hTri,hTri,sizeof(hyperTrial) *counter2, hipMemcpyHostToDevice);
  cout<<"copy to device again"<<endl;
  int newSize = curUser.TAS.size();
  cout<<"new TAS SIZE is: "<<newSize<<"; old TAS size is"<<tasSize<<endl;
  hipDeviceSynchronize();
  while(newSize>0.10*tasSize)
  {
    
    newSize=curUser.TAS.size();
    cout<<counter<<" ||||||| "<<counter2<<" ||||||||||  "<<newSize<<endl;
    int tasInd=0;
    for(string t:curUser.TAS)
    {
        stringstream tt(t);
        float x,y;
        tt>>x;
        tt>>y;
        TAS[tasInd++]=x;
        TAS[tasInd++]=y;
    }
    hipFree(d_TAS);
    hipMalloc((void**)&d_TAS, sizeof(float) *newSize*2);
    hipMemcpy(d_TAS, TAS, sizeof(float) *newSize*2, hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    goOver3<<<2048,256>>>(counter, d_eTri, d_TAS, newSize*2);
    hipDeviceSynchronize();
     
    
    
    
    
    hipDeviceSynchronize();
    goOver2<<<2048,256>>>(counter2, d_hTri, d_TAS, newSize*2);
    hipDeviceSynchronize();
    hipMemcpy(eTri,d_eTri,sizeof(ellipseTrial) *counter,hipMemcpyDeviceToHost); 
    hipMemcpy(hTri,d_hTri,sizeof(hyperTrial) *counter2,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    //free(cTri);
    //free(hTri);
    //cTri=(twoCycleTrial*)malloc(sizeof(twoCycleTrial)*counter);
    //hTri=(hyperTrial*)malloc(sizeof(hyperTrial)*counter2);
    
    
    res+=findBestTry2(eTri,hTri, counter, counter2, curUser.TAS);
  }
  hipDeviceSynchronize();
  hipFree(d_eTri);
  hipFree(d_hTri);
  hipFree(d_TAS);
  free(TAS);
  free(eTri); 
  free(hTri);
  
  //cout<<"sending back result!!!!!!!!!!!!!!!"<<endl;
    
  return res;


}

void writeToFile(string msg, int sn, int id, int mode,string network)
{

  string filename =  "../results/"+network+"_d/"+ to_string(sn)+"_"+to_string(id)+"_";
  if(mode==0)filename+="withoutEllipse";
  else filename+="withEllipse";
  ofstream resFile;
  resFile.open(filename);
  resFile<<msg;  
  resFile.close();
}

int main(int argc , char *argv[])
{  
  srand(time(NULL));
  User curUser;
  int mode;
  while(1){
    cout << "type in the mode: "<<endl;
    cout << "1: create random network"<<endl;
    cout << "2: find route msg for specific random network"<<endl;
    
    
    cin >> mode;
    if(mode ==1){
      int nn, an , rr;
      cout<< "Please type in number of nodes"<<endl;
      cin >> nn;
      cout<< "please type in maximum number of anchor nodes"<<endl;
      cin >> an;
      cout<< "please type in the radio range"<<endl;
      cin >> rr;
      string nid = curUser.randomNetwork(nn,an,rr);
      cout<< "created random network: "<<nid<<endl;
    }
    else if(mode == 2){
      string network;
      cout<< "Please type in the network ID"<<endl;
      cin >> network;
      curUser.loadNetwork(network);
      short hv[nodeSize][anchorSize];
      curUser.getHopInfo(hv);
      
      vector<twoCycleTrial> cycleTrials=curUser.findTwoCycleTrial(hv);
      cout<<"number of cycleTrials: "<<cycleTrials.size()<<endl;
      vector<ellipseTrial> ellipseTrials=curUser.findEllipseTrial(hv);
      cout<<"number of ellipseTrials: "<<ellipseTrials.size()<<endl;
      
      vector<hyperTrial> hyperTrials=curUser.findHyperTrial(hv);
      cout<<"number of hyperTrial: "<<hyperTrials.size()<<endl; 
      for(int i=180; i>=20; i--)
      {
        for(int j=0; j<20; j++)
        {
          curUser.updateStroke(i,j,50);
          cout<<"curUser TAS: "<<curUser.TAS.size()<<endl;
          string res = getRoutingMSG_without_ellipse(curUser,cycleTrials,hyperTrials);
          writeToFile(res,i,j,0,network);
          cout<<i<<"_"<<j<<" result without ellipse: "<<res<<endl;
          
          //string res2 = getRoutingMSG_with_ellipse(curUser,ellipseTrials,hyperTrials);
          //writeToFile(res2,i,j,1,network);
          //cout<<i<<"_"<<j<<" result with ellipse: "<< res2<<endl;
        }
      }
    
    }
    else{
      cout<<" please type 1 or 2"<<endl;
    
    }
  }


	return 0;
}



